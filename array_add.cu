#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel function to add two arrays
__global__ void arrayAdd(int *a, int *b, int *c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N)
        c[idx] = a[idx] + b[idx];
}

int main(){
    const int N = 10;
    int size = N * sizeof(int);

    // Allocate host arrays
    int h_a[N], h_b[N], h_c[N];

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        h_a[i] = i;
        h_b[i] = i * i;
    }

    // Allocate device arrays
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data to GPU
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel (1 block, N threads)
    arrayAdd<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print result
    printf("Result:\n");
    for(int i = 0; i < N; i++){
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
